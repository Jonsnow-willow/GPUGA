/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The class defining the simulation box.
------------------------------------------------------------------------------*/


#include "box.cuh"
#include "error.cuh"
#include "common.cuh"


void Box::read_file(char* input_dir, int Nc)
{
    print_line_1();
    printf("Started reading box.in.\n");
    print_line_2();
    char file_box[200];
    strcpy(file_box, input_dir);
    strcat(file_box, "/box.in");
    FILE *fid_box = my_fopen(file_box, "r");

    CHECK(hipMallocManaged((void**)&triclinic, sizeof(int) * Nc));
    CHECK(hipMallocManaged((void**)&h, sizeof(float) * Nc * 18));
    MY_MALLOC(cpu_pe_ref, float, Nc);
    MY_MALLOC(cpu_sxx_ref, float, Nc);
    MY_MALLOC(cpu_syy_ref, float, Nc);
    MY_MALLOC(cpu_szz_ref, float, Nc);
    
    potential_square_sum = 0.0;
    virial_square_sum = 0.0;
    for (int n = 0; n < Nc; ++n)
    {
        float *h_local = h + n * 18; // define a local pointer

        int count = fscanf(fid_box, "%d%f%f%f%f", 
            &triclinic[n], &cpu_pe_ref[n], &cpu_sxx_ref[n],
            &cpu_syy_ref[n], &cpu_szz_ref[n]);

        if (n >= NC_FORCE)
        {
            float energy = cpu_pe_ref[n] + 4.63 * MAX_ATOM_NUMBER;
            potential_square_sum += energy * energy;
            virial_square_sum += cpu_sxx_ref[n] * cpu_sxx_ref[n]
                               + cpu_syy_ref[n] * cpu_syy_ref[n]
                               + cpu_szz_ref[n] * cpu_szz_ref[n];
        }

        if (count != 5) print_error("Reading error for box.in.\n");
        if (triclinic[n] == 0) printf("orthogonal %g %g %g %g\n", 
            cpu_pe_ref[n], cpu_sxx_ref[n], cpu_syy_ref[n], cpu_szz_ref[n]);
        else if (triclinic[n] == 1) printf("triclinic %g %g %g %g\n", 
            cpu_pe_ref[n], cpu_sxx_ref[n], cpu_syy_ref[n], cpu_szz_ref[n]);
        else print_error("Invalid box type.\n");

        if (triclinic[n] == 1)
        {
            float ax, ay, az, bx, by, bz, cx, cy, cz;
            int count = fscanf(fid_box, "%f%f%f%f%f%f%f%f%f",
                &ax, &ay, &az, &bx, &by, &bz, &cx, &cy, &cz);
            if (count != 9) print_error("reading error for box.in.\n");
            h_local[0] = ax; h_local[3] = ay; h_local[6] = az;
            h_local[1] = bx; h_local[4] = by; h_local[7] = bz;
            h_local[2] = cx; h_local[5] = cy; h_local[8] = cz;
            get_inverse(triclinic[n], h_local);
            for (int k = 0; k < 9; ++k) printf("%g ", h_local[k]);
            printf("\n");
        }
        else
        {
            float lx, ly, lz;
            int count = fscanf(fid_box, "%f%f%f", &lx, &ly, &lz);
            if (count != 3) print_error("reading error for box.in.\n");
            h_local[0] = lx; h_local[1] = ly; h_local[2] = lz;
            printf("%g %g %g\n", lx, ly, lz);
        }
    }
    fclose(fid_box);

    int memory = sizeof(float) * Nc;
    CHECK(hipMalloc((void**)&pe_ref, memory));
    CHECK(hipMalloc((void**)&sxx_ref, memory));
    CHECK(hipMalloc((void**)&syy_ref, memory));
    CHECK(hipMalloc((void**)&szz_ref, memory));
    CHECK(hipMemcpy(pe_ref, cpu_pe_ref, memory, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(sxx_ref, cpu_sxx_ref, memory, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(syy_ref, cpu_syy_ref, memory, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(szz_ref, cpu_szz_ref, memory, hipMemcpyHostToDevice));
}  


Box::~Box(void)
{
    CHECK(hipFree(triclinic));
    CHECK(hipFree(h));
    MY_FREE(cpu_pe_ref);
    MY_FREE(cpu_sxx_ref);
    MY_FREE(cpu_syy_ref);
    MY_FREE(cpu_szz_ref);
    CHECK(hipFree(pe_ref)); 
    CHECK(hipFree(sxx_ref));
    CHECK(hipFree(syy_ref));
    CHECK(hipFree(szz_ref));
}


float Box::get_volume(int triclinic, float *cpu_h)
{
    float volume;
    if (triclinic)
    {
        volume = cpu_h[0] * (cpu_h[4]*cpu_h[8] - cpu_h[5]*cpu_h[7])
               + cpu_h[1] * (cpu_h[5]*cpu_h[6] - cpu_h[3]*cpu_h[8])
               + cpu_h[2] * (cpu_h[3]*cpu_h[7] - cpu_h[4]*cpu_h[6]);
    }
    else
    {
        volume = cpu_h[0] * cpu_h[1] * cpu_h[2];
    }
    return volume;
}


void Box::get_inverse(int triclinic, float *cpu_h)
{
    cpu_h[9]  = cpu_h[4]*cpu_h[8] - cpu_h[5]*cpu_h[7];
    cpu_h[10] = cpu_h[2]*cpu_h[7] - cpu_h[1]*cpu_h[8];
    cpu_h[11] = cpu_h[1]*cpu_h[5] - cpu_h[2]*cpu_h[4];
    cpu_h[12] = cpu_h[5]*cpu_h[6] - cpu_h[3]*cpu_h[8];
    cpu_h[13] = cpu_h[0]*cpu_h[8] - cpu_h[2]*cpu_h[6];
    cpu_h[14] = cpu_h[2]*cpu_h[3] - cpu_h[0]*cpu_h[5];
    cpu_h[15] = cpu_h[3]*cpu_h[7] - cpu_h[4]*cpu_h[6];
    cpu_h[16] = cpu_h[1]*cpu_h[6] - cpu_h[0]*cpu_h[7];
    cpu_h[17] = cpu_h[0]*cpu_h[4] - cpu_h[1]*cpu_h[3];
    float volume = get_volume(triclinic, cpu_h);
    for (int n = 9; n < 18; n++)
    {
        cpu_h[n] /= volume;
    }
}


