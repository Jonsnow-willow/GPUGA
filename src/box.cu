/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The class defining the simulation box.
------------------------------------------------------------------------------*/


#include "box.cuh"
#include "error.cuh"


Box::~Box(void)
{
    CHECK(hipFree(triclinic));
    CHECK(hipFree(h));
    CHECK(hipFree(pe_ref)); 
    CHECK(hipFree(sxx_ref));
    CHECK(hipFree(syy_ref));
    CHECK(hipFree(szz_ref));
}


float Box::get_volume(int triclinic, float *cpu_h)
{
    float volume;
    if (triclinic)
    {
        volume = cpu_h[0] * (cpu_h[4]*cpu_h[8] - cpu_h[5]*cpu_h[7])
               + cpu_h[1] * (cpu_h[5]*cpu_h[6] - cpu_h[3]*cpu_h[8])
               + cpu_h[2] * (cpu_h[3]*cpu_h[7] - cpu_h[4]*cpu_h[6]);
    }
    else
    {
        volume = cpu_h[0] * cpu_h[1] * cpu_h[2];
    }
    return volume;
}


void Box::get_inverse(int triclinic, float *cpu_h)
{
    cpu_h[9]  = cpu_h[4]*cpu_h[8] - cpu_h[5]*cpu_h[7];
    cpu_h[10] = cpu_h[2]*cpu_h[7] - cpu_h[1]*cpu_h[8];
    cpu_h[11] = cpu_h[1]*cpu_h[5] - cpu_h[2]*cpu_h[4];
    cpu_h[12] = cpu_h[5]*cpu_h[6] - cpu_h[3]*cpu_h[8];
    cpu_h[13] = cpu_h[0]*cpu_h[8] - cpu_h[2]*cpu_h[6];
    cpu_h[14] = cpu_h[2]*cpu_h[3] - cpu_h[0]*cpu_h[5];
    cpu_h[15] = cpu_h[3]*cpu_h[7] - cpu_h[4]*cpu_h[6];
    cpu_h[16] = cpu_h[1]*cpu_h[6] - cpu_h[0]*cpu_h[7];
    cpu_h[17] = cpu_h[0]*cpu_h[4] - cpu_h[1]*cpu_h[3];
    float volume = get_volume(triclinic, cpu_h);
    for (int n = 9; n < 18; n++)
    {
        cpu_h[n] /= volume;
    }
}


