/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The class defining the simulation box.
------------------------------------------------------------------------------*/


#include "box.cuh"
#include "error.cuh"


void Box::read_file(char* input_dir, int Nc)
{
    print_line_1();
    printf("Started reading box.in.\n");
    print_line_2();
    char file_box[200];
    strcpy(file_box, input_dir);
    strcat(file_box, "/box.in");
    FILE *fid_box = my_fopen(file_box, "r");

    MY_MALLOC(cpu_pe_ref, double, Nc);
    MY_MALLOC(cpu_h, double, 18 * Nc);
    pe_ref_square_sum = 0.0;
    for (int n = 0; n < Nc; ++n)
    {
        double *h_local = cpu_h + n * 18; // define a local pointer

        int count = fscanf(fid_box, "%d%lf", &triclinic, &cpu_pe_ref[n]);
        if (count != 2) print_error("Reading error for box.in.\n");
        if (triclinic == 0) printf("orthogonal %g\n", cpu_pe_ref[n]);
        else if (triclinic == 1) printf("triclinic %g\n", cpu_pe_ref[n]);
        else print_error("Invalid box type.\n");

        pe_ref_square_sum += cpu_pe_ref[n] * cpu_pe_ref[n];

        if (triclinic == 1)
        {
            double ax, ay, az, bx, by, bz, cx, cy, cz;
            int count = fscanf(fid_box, "%lf%lf%lf%lf%lf%lf%lf%lf%lf",
                &ax, &ay, &az, &bx, &by, &bz, &cx, &cy, &cz);
            if (count != 9) print_error("reading error for box.in.\n");
            h_local[0] = ax; h_local[3] = ay; h_local[6] = az;
            h_local[1] = bx; h_local[4] = by; h_local[7] = bz;
            h_local[2] = cx; h_local[5] = cy; h_local[8] = cz;
            get_inverse(h_local);
            for (int k = 0; k < 9; ++k) printf("%g ", h_local[k]);
            printf("\n");
        }
        else
        {
            double lx, ly, lz;
            int count = fscanf(fid_box, "%lf%lf%lf", &lx, &ly, &lz);
            if (count != 3) print_error("reading error for box.in.\n");
            h_local[0] = lx; h_local[1] = ly; h_local[2] = lz;
            printf("%g %g %g\n", lx, ly, lz);
        }
    }
    fclose(fid_box);

    int memory = sizeof(double) * Nc * 18;
    CHECK(hipMalloc((void**)&h, memory));
    CHECK(hipMemcpy(h, cpu_h, memory, hipMemcpyHostToDevice));
    memory = sizeof(double) * Nc;
    CHECK(hipMalloc((void**)&pe_ref, memory));
    CHECK(hipMemcpy(pe_ref, cpu_pe_ref, memory, hipMemcpyHostToDevice));
}  


Box::~Box(void)
{
    MY_FREE(cpu_h);
    CHECK(hipFree(h));
    MY_FREE(cpu_pe_ref);
    CHECK(hipFree(pe_ref)); 
}


double Box::get_volume(double *cpu_h)
{
    double volume;
    if (triclinic)
    {
        volume = cpu_h[0] * (cpu_h[4]*cpu_h[8] - cpu_h[5]*cpu_h[7])
               + cpu_h[1] * (cpu_h[5]*cpu_h[6] - cpu_h[3]*cpu_h[8])
               + cpu_h[2] * (cpu_h[3]*cpu_h[7] - cpu_h[4]*cpu_h[6]);
    }
    else
    {
        volume = cpu_h[0] * cpu_h[1] * cpu_h[2];
    }
    return volume;
}


void Box::get_inverse(double *cpu_h)
{
    cpu_h[9]  = cpu_h[4]*cpu_h[8] - cpu_h[5]*cpu_h[7];
    cpu_h[10] = cpu_h[2]*cpu_h[7] - cpu_h[1]*cpu_h[8];
    cpu_h[11] = cpu_h[1]*cpu_h[5] - cpu_h[2]*cpu_h[4];
    cpu_h[12] = cpu_h[5]*cpu_h[6] - cpu_h[3]*cpu_h[8];
    cpu_h[13] = cpu_h[0]*cpu_h[8] - cpu_h[2]*cpu_h[6];
    cpu_h[14] = cpu_h[2]*cpu_h[3] - cpu_h[0]*cpu_h[5];
    cpu_h[15] = cpu_h[3]*cpu_h[7] - cpu_h[4]*cpu_h[6];
    cpu_h[16] = cpu_h[1]*cpu_h[6] - cpu_h[0]*cpu_h[7];
    cpu_h[17] = cpu_h[0]*cpu_h[4] - cpu_h[1]*cpu_h[3];
    double volume = get_volume(cpu_h);
    for (int n = 9; n < 18; n++)
    {
        cpu_h[n] /= volume;
    }
}


