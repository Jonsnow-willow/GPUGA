#include "hip/hip_runtime.h"
/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
find the neighbor list
------------------------------------------------------------------------------*/


#include "fitness.cuh"
#include "mic.cuh"
#include "error.cuh"
#define BLOCK_SIZE 256


static __global__ void gpu_find_neighbor
(
    int triclinic, int pbc_x, int pbc_y, int pbc_z, 
    int N, int *Na, int *Na_sum,
    double cutoff_square, const double* __restrict__ box, 
    int *NN, int *NL, double *x, double *y, double *z
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
        double x1 = x[n1];  
        double y1 = y[n1];  
        double z1 = z[n1];
        int count = 0;
        for (int n2 = N1; n2 < N2; ++n2)
        { 
            if (n2 == n1) { continue; }
            double x12 = x[n2]-x1; 
            double y12 = y[n2]-y1; 
            double z12 = z[n2]-z1;
            dev_apply_mic(triclinic, pbc_x, pbc_y, pbc_z, box, x12, y12, z12);
            double distance_square = x12 * x12 + y12 * y12 + z12 * z12;
            if (distance_square < cutoff_square) { NL[count++ * N + n1] = n2; }
        }
        NN[n1] = count;
    }
}


void Fitness::find_neighbor(void)
{
    double rc2 = cutoff * cutoff;
    gpu_find_neighbor<<<Nc, BLOCK_SIZE>>>
    (
        box.triclinic, box.pbc_x, box.pbc_y, box.pbc_z,
        N, Na, Na_sum, rc2, box.h, NN, NL, x, y, z
    );
    CUDA_CHECK_KERNEL
}


