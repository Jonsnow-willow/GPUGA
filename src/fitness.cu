#include "hip/hip_runtime.h"
/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Get the fitness
------------------------------------------------------------------------------*/


#include "fitness.cuh"
#include "neighbor.cuh"
#include "error.cuh"
#include "read_file.cuh"


Fitness::Fitness(char* input_dir)
{
    read_potential(input_dir);
    read_weight(input_dir);
    read_xyz_in(input_dir);
    box.read_file(input_dir, Nc);
    neighbor.compute(Nc, N, Na, Na_sum, x, y, z, &box);
    potential.initialize(N, MAX_ATOM_NUMBER);
    MY_MALLOC(error_cpu, float, Nc);
    CHECK(hipMalloc((void**)&error_gpu, sizeof(float) * Nc));
}


Fitness::~Fitness(void)
{
    hipFree(Na);
    hipFree(Na_sum);
    hipFree(type);
    hipFree(x);
    hipFree(y);
    hipFree(z);
    hipFree(fx_ref);
    hipFree(fy_ref);
    hipFree(fz_ref);
    hipFree(pe);
    hipFree(sxx);
    hipFree(syy);
    hipFree(szz);
    hipFree(fx);
    hipFree(fy);
    hipFree(fz);
    CHECK(hipFree(error_gpu));
    MY_FREE(error_cpu);
    MY_FREE(parameters_min);
    MY_FREE(parameters_max);
}


void Fitness::read_xyz_in(char* input_dir)
{
    print_line_1();
    printf("Started reading xyz.in.\n");
    print_line_2();
    char file_xyz[200];
    strcpy(file_xyz, input_dir);
    strcat(file_xyz, "/xyz.in");
    FILE *fid_xyz = my_fopen(file_xyz, "r");
    read_Nc(fid_xyz);
    read_Na(fid_xyz);
    read_xyz(fid_xyz);
    fclose(fid_xyz);
}


void Fitness::read_weight(char* input_dir)
{
    print_line_1();
    printf("Started reading weight.in.\n");
    print_line_2();

    char file_weight[200];
    strcpy(file_weight, input_dir);
    strcat(file_weight, "/weight.in");
    FILE *fid = my_fopen(file_weight, "r");

    int count = fscanf
    (
        fid, "%f%f%f", &weight.force, &weight.energy, &weight.stress
    );
    if (count != 3) print_error("Reading error for weight.in.\n");

    fclose(fid);

    if (weight.force < 0)
    {
        print_error("weight.force should >= 0\n");
    }
    else
    {
        printf("weight.force = %g.\n", weight.force);
    }

    if (weight.energy < 0)
    {
        print_error("weight.energy should >= 0\n");
    }
    else
    {
        printf("weight.energy = %g.\n", weight.energy);
    }

    if (weight.stress < 0)
    {
        print_error("weight.stress should >= 0\n");
    }
    else
    {
        printf("weight.stress = %g.\n", weight.stress);
    }
}


void Fitness::read_Nc(FILE* fid)
{
    int count = fscanf(fid, "%d%d", &Nc, &NC_FORCE);
    if (count != 2) print_error("Reading error for xyz.in.\n");

    if (Nc < 2)
    {
        print_error("Number of configurations should >= 2\n");
    }

    if (NC_FORCE < 1)
    {
        print_error("Number of force configurations should >= 1\n");
    }
    else if (NC_FORCE > Nc - 1)
    {
        print_error("Number of potential configurations should >= 1\n");
    }

    printf("Number of configurations is %d:\n", Nc);
    printf("    %d force configurations;\n", NC_FORCE);
    printf("    %d energy and virial configurations.\n", Nc - NC_FORCE);
}


void Fitness::read_Na(FILE* fid)
{ 
    CHECK(hipMallocManaged((void**)&Na, sizeof(int) * Nc));
    CHECK(hipMallocManaged((void**)&Na_sum, sizeof(int) * Nc));

    N = 0;
    MAX_ATOM_NUMBER = 0;

    for (int nc = 0; nc < Nc; ++nc)
    {
        Na_sum[nc] = 0;
    }

    for (int nc = 0; nc < Nc; ++nc)
    {
        int count = fscanf(fid, "%d", &Na[nc]);

        if (count != 1)
        {
            print_error("Reading error for xyz.in.\n");
        }

        N += Na[nc];
        if (Na[nc] > MAX_ATOM_NUMBER)
        {
            MAX_ATOM_NUMBER = Na[nc];
        }

        if (Na[nc] < 1)
        {
            print_error("Number of atoms %d should >= 1\n");
        }
    }

    for (int nc = 1; nc < Nc; ++nc)
    {
        Na_sum[nc] = Na_sum[nc-1] + Na[nc-1];
    }

    // get the total number of atoms in force configurations
    N_force = 0;
    for (int nc = 0; nc < NC_FORCE; ++nc)
    {
        N_force += Na[nc];
    }
    printf("Total number of atoms is %d:\n", N);
    printf("    %d in force configurations;\n", N_force);
    printf("    %d in energy and virial configurations.\n", N - N_force);
} 


void Fitness::read_xyz(FILE* fid)
{
    int m1 = sizeof(int) * N;
    int m2 = sizeof(float) * N;

    CHECK(hipMallocManaged((void**)&type, m1));
    CHECK(hipMallocManaged((void**)&x, m2));
    CHECK(hipMallocManaged((void**)&y, m2));
    CHECK(hipMallocManaged((void**)&z, m2));
    CHECK(hipMallocManaged((void**)&fx, m2));
    CHECK(hipMallocManaged((void**)&fy, m2));
    CHECK(hipMallocManaged((void**)&fz, m2));
    CHECK(hipMallocManaged((void**)&fx_ref, m2));
    CHECK(hipMallocManaged((void**)&fy_ref, m2));
    CHECK(hipMallocManaged((void**)&fz_ref, m2));
    CHECK(hipMallocManaged((void**)&pe, m2));
    CHECK(hipMallocManaged((void**)&sxx, m2));
    CHECK(hipMallocManaged((void**)&syy, m2));
    CHECK(hipMallocManaged((void**)&szz, m2));

    num_types = 0;
    force_square_sum = 0.0;
    for (int n = 0; n < N; n++)
    {
        int count = fscanf(fid, "%d%f%f%f%f%f%f", 
            &(type[n]), &(x[n]), &(y[n]), &(z[n]),
            &(fx_ref[n]), &(fy_ref[n]), &(fz_ref[n]));
        if (count != 7) { print_error("reading error for xyz.in.\n"); }
        if (type[n] > num_types) { num_types = type[n]; }
        if (n < NC_FORCE * MAX_ATOM_NUMBER)
        {
            force_square_sum += fx_ref[n] * fx_ref[n]
                              + fy_ref[n] * fy_ref[n]
                              + fz_ref[n] * fz_ref[n];
        }
    }
    num_types++;
}


void Fitness::read_potential(char* input_dir)
{
    print_line_1();
    printf("Started reading potential.in.\n");
    print_line_2();
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/potential.in");
    FILE* fid = my_fopen(file, "r");
    int count = fscanf(fid, "%d", &number_of_variables);
    if (count != 1) { print_error("reading error for potential.in."); }
    printf("number of variables = %d\n", number_of_variables);

    MY_MALLOC(parameters_min, float, number_of_variables);
    MY_MALLOC(parameters_max, float, number_of_variables);

    char name[20];

    count = fscanf(fid, "%s%f", name, &neighbor.cutoff);
    if (count != 2) { print_error("reading error for potential.in."); }
    printf("cutoff for neighbor list is %f.\n", neighbor.cutoff);

    for (int n = 0; n <  number_of_variables; ++n)
    {
        count = fscanf
        (fid, "%s%f%f", name, &parameters_min[n], &parameters_max[n]);
        if (count != 3) { print_error("reading error for potential.in."); }
        printf("%15s%15g%15g\n", name, parameters_min[n], parameters_max[n]);
    }
    fclose(fid);
}


void Fitness::compute(int population_size, float* population, float* fitness)
{
    float *parameters;
    MY_MALLOC(parameters, float, number_of_variables);

    for (int n = 0; n < population_size; ++n)
    {
        float* individual = population + n * number_of_variables;
        for (int m = 0; m < number_of_variables; ++m)
        {
            float a = parameters_min[m];
            float b = parameters_max[m] - a;
            parameters[m] = a + b * individual[m];
        }
        potential.update_potential(parameters, num_types);
        potential.find_force
        (
            num_types, Nc, N, Na, Na_sum, MAX_ATOM_NUMBER, type, &box, &neighbor,
            x, y, z, fx, fy, fz, sxx, syy, szz, pe
        );
        fitness[n] = weight.energy * get_fitness_energy();
        fitness[n] += weight.stress * get_fitness_stress();
        fitness[n] += weight.force * get_fitness_force();
    }

    MY_FREE(parameters);
}


void Fitness::predict_energy_or_stress(FILE* fid, float* data, float* ref)
{
    for (int nc = NC_FORCE; nc < Nc; ++nc)
    {
        int offset = Na_sum[nc];
        float data_nc = 0.0;
        for (int m = 0; m < Na[nc]; ++m)
        {
            data_nc += data[offset + m];
        }
        fprintf(fid, "%25.15e%25.15e\n", data_nc, ref[nc]);
    }
}


void Fitness::predict(char* input_dir, float* elite)
{
    float *parameters;
    MY_MALLOC(parameters, float, number_of_variables);
    for (int m = 0; m < number_of_variables; ++m)
    {
        float a = parameters_min[m];
        float b = parameters_max[m] - a;
        parameters[m] = a + b * elite[m];
    }
    potential.update_potential(parameters, num_types);
    potential.find_force
    (
        num_types, Nc, N, Na, Na_sum, MAX_ATOM_NUMBER, type, &box, &neighbor,
        x, y, z, fx, fy, fz, sxx, syy, szz, pe
    );
    MY_FREE(parameters);

    CHECK(hipDeviceSynchronize()); // needed for CC < 6.0

    char file_force[200];
    strcpy(file_force, input_dir);
    strcat(file_force, "/force.out");
    FILE* fid_force = my_fopen(file_force, "w");
    for (int n = 0; n < N_force; ++n)
    {
        fprintf
        (
            fid_force, "%25.15e%25.15e%25.15e%25.15e%25.15e%25.15e\n", 
            fx[n], fy[n], fz[n], fx_ref[n], fy_ref[n], fz_ref[n]
        );
    }
    fclose(fid_force);

    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/prediction.out");
    FILE* fid_prediction = my_fopen(file, "w");
    predict_energy_or_stress(fid_prediction, pe, box.pe_ref);
    predict_energy_or_stress(fid_prediction, sxx, box.sxx_ref);
    predict_energy_or_stress(fid_prediction, syy, box.syy_ref);
    predict_energy_or_stress(fid_prediction, szz, box.szz_ref);
    fclose(fid_prediction);
}


static __global__ void gpu_sum_force_error
(
    int N, float *g_fx, float *g_fy, float *g_fz, 
    float *g_fx_ref, float *g_fy_ref, float *g_fz_ref, float *g_error
)
{
    int tid = threadIdx.x;
    int number_of_rounds = (N - 1) / blockDim.x + 1; 
    extern __shared__ float s_error[];
    s_error[tid] = 0.0;
    for (int round = 0; round < number_of_rounds; ++round)
    {
        int n = tid + round * blockDim.x;
        if (n < N)
        {
            float dx = g_fx[n] - g_fx_ref[n];
            float dy = g_fy[n] - g_fy_ref[n];
            float dz = g_fz[n] - g_fz_ref[n];
            s_error[tid] += dx * dx + dy * dy + dz * dz;
        }
    }

    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1)
    {
        if (tid < offset) { s_error[tid] += s_error[tid + offset]; }
        __syncthreads();
    }

    for (int offset = 32; offset > 0; offset >>= 1)
    {
        if (tid < offset) { s_error[tid] += s_error[tid + offset]; }
        __syncwarp();
    }

    if (tid ==  0) { g_error[0] = s_error[0]; }
}


float Fitness::get_fitness_force(void)
{
    gpu_sum_force_error<<<1, 512, sizeof(float) * 512>>>
    (N_force, fx, fy, fz, fx_ref, fy_ref, fz_ref, error_gpu);
    CHECK(hipMemcpy(error_cpu, error_gpu, sizeof(float), 
        hipMemcpyDeviceToHost));
    return sqrt(error_cpu[0] / force_square_sum);
}


static __global__ void gpu_sum_pe_error
(int *g_Na, int *g_Na_sum, float *g_pe, float *g_pe_ref, float *error_gpu)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int Na = g_Na[bid];
    int offset = g_Na_sum[bid];
    extern __shared__ float s_pe[];
    s_pe[tid] = 0.0;
    if (tid < Na)
    {
        int n = offset + tid; // particle index
        s_pe[tid] += g_pe[n];
    }
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1)
    {
        if (tid < offset) { s_pe[tid] += s_pe[tid + offset]; }
        __syncthreads();
    }

    for (int offset = 32; offset > 0; offset >>= 1)
    {
        if (tid < offset) { s_pe[tid] += s_pe[tid + offset]; }
        __syncwarp();
    }

    if (tid == 0)
    {
        float diff = s_pe[0] - g_pe_ref[bid];
        error_gpu[bid] = diff * diff;
    }
}


static int get_block_size(int max_num_atom)
{
    int block_size = 64;
    for (int n = 64; n < 1024; n <<= 1 )
    {
        if (max_num_atom > n)
        {
            block_size = n << 1;
        }
    }
    return block_size;
}


float Fitness::get_fitness_energy(void)
{
    int block_size = get_block_size(MAX_ATOM_NUMBER);
    gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>
    (Na, Na_sum, pe, box.pe_ref, error_gpu);
    int mem = sizeof(float) * Nc;
    CHECK(hipMemcpy(error_cpu, error_gpu, mem, hipMemcpyDeviceToHost));
    float error_ave = 0.0;
    for (int n = NC_FORCE; n < Nc; ++n)
    {
        error_ave += error_cpu[n];
    }
    return sqrt(error_ave / box.potential_square_sum);
}


float Fitness::get_fitness_stress(void)
{
    float error_ave = 0.0;
    int mem = sizeof(float) * Nc;
    int block_size = get_block_size(MAX_ATOM_NUMBER);

    gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>
    (Na, Na_sum, sxx, box.sxx_ref, error_gpu);
    CHECK(hipMemcpy(error_cpu, error_gpu, mem, hipMemcpyDeviceToHost));
    for (int n = NC_FORCE; n < Nc; ++n) {error_ave += error_cpu[n];}

    gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>
    (Na, Na_sum, syy, box.syy_ref, error_gpu);
    CHECK(hipMemcpy(error_cpu, error_gpu, mem, hipMemcpyDeviceToHost));
    for (int n = NC_FORCE; n < Nc; ++n) {error_ave += error_cpu[n];}

    gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>
    (Na, Na_sum, szz, box.szz_ref, error_gpu);
    CHECK(hipMemcpy(error_cpu, error_gpu, mem, hipMemcpyDeviceToHost));
    for (int n = NC_FORCE; n < Nc; ++n) {error_ave += error_cpu[n];}

    return sqrt(error_ave / box.virial_square_sum);
}


