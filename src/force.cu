#include "hip/hip_runtime.h"
/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Calculate force, energy, and stress
------------------------------------------------------------------------------*/


#include "fitness.cuh"
#include "mic.cuh"
#include "error.cuh"
#include "common.cuh"

//Easy labels for indexing
#define A      0
#define Q      1
#define LAMBDA 2
#define B      3
#define MU     4
#define B2     5
#define MU2    6
#define BETA   7
#define EN     8 // special name for n to avoid conflict
#define H      9
#define R1     10
#define R2     11
#define PI_FACTOR 12
#define MINUS_HALF_OVER_N 13


void Fitness::update_potential(double* potential_parameters)
{
    int n_entries = num_types * num_types * num_types;
    double a = potential_parameters[0];
    double q = potential_parameters[1];
    double lambda = potential_parameters[2];
    double b = potential_parameters[3];
    double mu = potential_parameters[4];
    double b2 = potential_parameters[5];
    double mu2 = potential_parameters[6];
    double beta = potential_parameters[7];
    double r1 = potential_parameters[8];
    double r2 = potential_parameters[9];
    double n = 1.0;
    double h = -1.0/3.0;


    for (int i = 0; i < n_entries; i++)
    {
        cpu_ters[i*NUM_PARAMS + A] = a;
        cpu_ters[i*NUM_PARAMS + B] = b;
        cpu_ters[i*NUM_PARAMS + LAMBDA] = lambda;
        cpu_ters[i*NUM_PARAMS + MU] = mu;
        cpu_ters[i*NUM_PARAMS + BETA] = beta;
        cpu_ters[i*NUM_PARAMS + EN] = n;
        cpu_ters[i*NUM_PARAMS + H] = h;
        cpu_ters[i*NUM_PARAMS + R1] = r1;
        cpu_ters[i*NUM_PARAMS + R2] = r2;
        cpu_ters[i*NUM_PARAMS + B2] = b2;
        cpu_ters[i*NUM_PARAMS + MU2] = mu2;
        cpu_ters[i*NUM_PARAMS + Q] = q;
        cpu_ters[i*NUM_PARAMS + PI_FACTOR] = PI / (r2 - r1);
        cpu_ters[i*NUM_PARAMS + MINUS_HALF_OVER_N] = - 0.5 / n;
    }
    int mem = sizeof(double) * n_entries * NUM_PARAMS;
    CHECK(hipMemcpy(ters, cpu_ters, mem, hipMemcpyHostToDevice));
}


static __device__ void find_fr_and_frp
(int i, const double* __restrict__ ters, double d12, double &fr, double &frp)
{
    double exp_factor = LDG(ters,i + A) * exp(- LDG(ters,i + LAMBDA) * d12);
    double d_inv = 1.0 / d12;
    fr = (1.0 + LDG(ters,i + Q) * d_inv) * exp_factor;
    frp = - LDG(ters, i + LAMBDA)*fr - LDG(ters, i + Q)*d_inv*d_inv*exp_factor;
}


static __device__ void find_fa_and_fap
(int i, const double* __restrict__ ters, double d12, double &fa, double &fap)
{
    fa  = LDG(ters, i + B) * exp(- LDG(ters, i + MU) * d12);
    fap = - LDG(ters, i + MU) * fa;
    double tmp =  LDG(ters, i + B2) * exp(- LDG(ters, i + MU2) * d12);
    fa += tmp;
    fap -= LDG(ters, i + MU2) * tmp;
}


static __device__ void find_fc_and_fcp
(int i, const double* __restrict__ ters, double d12, double &fc, double &fcp)
{
    if (d12 < LDG(ters, i + R1)){fc = 1.0; fcp = 0.0;}
    else if (d12 < LDG(ters, i + R2))
    {
        fc = 9.0/16.0 * cos(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1)))
           - 1.0/16 * cos(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1)) * 3.0)
           + 0.5;

        fcp = sin(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1)) * 3.0) 
                * LDG(ters, i + PI_FACTOR) * 3.0/ 16.0
                - sin(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1))) 
                * LDG(ters, i + PI_FACTOR) * 9.0 / 16.0;
    }
    else {fc  = 0.0; fcp = 0.0;}
}


static __device__ void find_fa
(int i, const double* __restrict__ ters, double d12, double &fa)
{
    fa = LDG(ters, i + B) * exp(- LDG(ters, i + MU) * d12);
    fa += LDG(ters, i + B2) * exp(- LDG(ters, i + MU2) * d12);
}


static __device__ void find_fc
(int i, const double* __restrict__ ters, double d12, double &fc)
{
    if (d12 < LDG(ters, i + R1)) {fc  = 1.0;}
    else if (d12 < LDG(ters, i + R2))
    {
        fc = 9.0/16.0 * cos(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1)))
           - 1.0/16 * cos(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1)) * 3.0)
           + 0.5;
    }
    else {fc  = 0.0;}
}


static __device__ void find_g_and_gp
(int i, const double* __restrict__ ters, double cos, double &g, double &gp)
{
    double temp = cos - LDG(ters, i + H);
    g  = temp * temp;
    gp = 2.0 * temp;
}


static __device__ void find_g
(int i, const double* __restrict__ ters, double cos, double &g)
{
    double temp = cos - LDG(ters, i + H);
    g  = temp * temp;
}


// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_tersoff_step1
(
    int number_of_particles, int *Na, int *Na_sum,
    const int* __restrict__ g_triclinic, 
    int num_types, int* g_neighbor_number, int* g_neighbor_list, int* g_type,
    const double* __restrict__ ters,
    const double* __restrict__ g_x,
    const double* __restrict__ g_y,
    const double* __restrict__ g_z,
    const double* __restrict__ g_box,
    double* g_b, double* g_bp
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
        int num_types2 = num_types * num_types;
        const double* __restrict__ h = g_box + 18 * blockIdx.x;
        int triclinic = LDG(g_triclinic, blockIdx.x);
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        double x1 = LDG(g_x, n1); 
        double y1 = LDG(g_y, n1); 
        double z1 = LDG(g_z, n1);
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];
            int type2 = g_type[n2];
            double x12  = LDG(g_x, n2) - x1;
            double y12  = LDG(g_y, n2) - y1;
            double z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(triclinic, h, x12, y12, z12);
            double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            double zeta = 0.0;
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];
                if (n3 == n2) { continue; } // ensure that n3 != n2
                int type3 = g_type[n3];
                double x13 = LDG(g_x, n3) - x1;
                double y13 = LDG(g_y, n3) - y1;
                double z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(triclinic, h, x13, y13, z13);
                double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12*d13);
                double fc_ijk_13, g_ijk;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                if (d13 > LDG(ters, ijk*NUM_PARAMS + R2)) {continue;}
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_g(ijk*NUM_PARAMS, ters, cos123, g_ijk);
                zeta += fc_ijk_13 * g_ijk;
            }
            double bzn, b_ijj;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            bzn = pow(LDG(ters, ijj*NUM_PARAMS + BETA) *
                zeta, LDG(ters, ijj*NUM_PARAMS + EN));
            b_ijj = 
                pow(1.0 + bzn, LDG(ters, ijj*NUM_PARAMS + MINUS_HALF_OVER_N));
            if (zeta < 1.0e-16) // avoid division by 0
            {
                g_b[i1 * number_of_particles + n1]  = 1.0;
                g_bp[i1 * number_of_particles + n1] = 0.0;
            }
            else
            {
                g_b[i1 * number_of_particles + n1]  = b_ijj;
                g_bp[i1 * number_of_particles + n1]
                    = - b_ijj * bzn * 0.5 / ((1.0 + bzn) * zeta);
            }
        }
    }
}


// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void find_force_tersoff_step2
(
    int number_of_particles, int *Na, int *Na_sum,
    const int* __restrict__ g_triclinic, 
    int num_types, int *g_neighbor_number, int *g_neighbor_list, int *g_type,
    const double* __restrict__ ters,
    const double* __restrict__ g_b,
    const double* __restrict__ g_bp,
    const double* __restrict__ g_x,
    const double* __restrict__ g_y,
    const double* __restrict__ g_z,
    const double* __restrict__ g_box,
    double *g_potential, double *g_f12x, double *g_f12y, double *g_f12z
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
        int num_types2 = num_types * num_types;
        const double* __restrict__ h = g_box + 18 * blockIdx.x;
        int triclinic = LDG(g_triclinic, blockIdx.x);
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        double x1 = LDG(g_x, n1); 
        double y1 = LDG(g_y, n1); 
        double z1 = LDG(g_z, n1);
        double pot_energy = 0.0;
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];
            int type2 = g_type[n2];

            double x12  = LDG(g_x, n2) - x1;
            double y12  = LDG(g_y, n2) - y1;
            double z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(triclinic, h, x12, y12, z12);
            double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            double d12inv = 1.0 / d12;
            double fc_ijj_12, fcp_ijj_12;
            double fa_ijj_12, fap_ijj_12, fr_ijj_12, frp_ijj_12;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            find_fc_and_fcp(ijj*NUM_PARAMS, ters, d12, fc_ijj_12, fcp_ijj_12);
            find_fa_and_fap(ijj*NUM_PARAMS, ters, d12, fa_ijj_12, fap_ijj_12);
            find_fr_and_frp(ijj*NUM_PARAMS, ters, d12, fr_ijj_12, frp_ijj_12);

            // (i,j) part
            double b12 = LDG(g_b, index);
            double factor3=(fcp_ijj_12*(fr_ijj_12-b12*fa_ijj_12)+
                            fc_ijj_12*(frp_ijj_12-b12*fap_ijj_12))*d12inv;
            double f12x = x12 * factor3 * 0.5;
            double f12y = y12 * factor3 * 0.5;
            double f12z = z12 * factor3 * 0.5;

            // accumulate potential energy
            pot_energy += fc_ijj_12 * (fr_ijj_12 - b12 * fa_ijj_12) * 0.5;

            // (i,j,k) part
            double bp12 = LDG(g_bp, index);
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int index_2 = n1 + number_of_particles * i2;
                int n3 = g_neighbor_list[index_2];
                if (n3 == n2) { continue; }
                int type3 = g_type[n3];
                double x13 = LDG(g_x, n3) - x1;
                double y13 = LDG(g_y, n3) - y1;
                double z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(triclinic, h, x13, y13, z13);
                double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                double fc_ikk_13, fc_ijk_13, fa_ikk_13, fc_ikj_12, fcp_ikj_12;
                int ikj = type1 * num_types2 + type3 * num_types + type2;
                int ikk = type1 * num_types2 + type3 * num_types + type3;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                find_fc(ikk*NUM_PARAMS, ters, d13, fc_ikk_13);
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_fa(ikk*NUM_PARAMS, ters, d13, fa_ikk_13);
                find_fc_and_fcp(ikj*NUM_PARAMS, ters, d12,
                                	fc_ikj_12, fcp_ikj_12);
                double bp13 = LDG(g_bp, index_2);
                double one_over_d12d13 = 1.0 / (d12 * d13);
                double cos123 = (x12*x13 + y12*y13 + z12*z13)*one_over_d12d13;
                double cos123_over_d12d12 = cos123*d12inv*d12inv;
                double g_ijk, gp_ijk;
                find_g_and_gp(ijk*NUM_PARAMS, ters, cos123, g_ijk, gp_ijk);

                double g_ikj, gp_ikj;
                find_g_and_gp(ikj*NUM_PARAMS, ters, cos123, g_ikj, gp_ikj);

                // derivatives with cosine
                double dc=-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*gp_ijk+
                        -fc_ikj_12*bp13*fa_ikk_13*fc_ikk_13*gp_ikj;
                // derivatives with rij
                double dr=(-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*g_ijk +
                  (-fcp_ikj_12*bp13*fa_ikk_13*g_ikj +
                  fc_ikj_12*bp13*fa_ikk_13*g_ikj)*fc_ikk_13)*d12inv;
                double cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
                f12x += (x12 * dr + dc * cos_d)*0.5;
                cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
                f12y += (y12 * dr + dc * cos_d)*0.5;
                cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
                f12z += (z12 * dr + dc * cos_d)*0.5;
            }
            g_f12x[index] = f12x; g_f12y[index] = f12y; g_f12z[index] = f12z;
        }
        // save potential
        g_potential[n1] = pot_energy;
    }
}


static __global__ void find_force_tersoff_step3
(
    int number_of_particles, int *Na, int *Na_sum,
    const int* __restrict__ g_triclinic,
    int *g_neighbor_number, int *g_neighbor_list,
    const double* __restrict__ g_f12x,
    const double* __restrict__ g_f12y,
    const double* __restrict__ g_f12z,
    const double* __restrict__ g_x,
    const double* __restrict__ g_y,
    const double* __restrict__ g_z,
    const double* __restrict__ g_box,
    double *g_fx, double *g_fy, double *g_fz,
    double *g_sx, double *g_sy, double *g_sz
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
        double s_fx = 0.0; // force_x
        double s_fy = 0.0; // force_y
        double s_fz = 0.0; // force_z
        double s_sx = 0.0; // virial_stress_x
        double s_sy = 0.0; // virial_stress_y
        double s_sz = 0.0; // virial_stress_z
        const double* __restrict__ h = g_box + 18 * blockIdx.x;
        int triclinic = LDG(g_triclinic, blockIdx.x);
        int neighbor_number = g_neighbor_number[n1];
        double x1 = LDG(g_x, n1); 
        double y1 = LDG(g_y, n1); 
        double z1 = LDG(g_z, n1);

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];
            int neighbor_number_2 = g_neighbor_number[n2];

            double x12  = LDG(g_x, n2) - x1;
            double y12  = LDG(g_y, n2) - y1;
            double z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(triclinic, h, x12, y12, z12);

            double f12x = LDG(g_f12x, index);
            double f12y = LDG(g_f12y, index);
            double f12z = LDG(g_f12z, index);
            int offset = 0;
            for (int k = 0; k < neighbor_number_2; ++k)
            {
                if (n1 == g_neighbor_list[n2 + number_of_particles * k])
                { offset = k; break; }
            }
            index = offset * number_of_particles + n2;
            double f21x = LDG(g_f12x, index);
            double f21y = LDG(g_f12y, index);
            double f21z = LDG(g_f12z, index);

            // per atom force
            s_fx += f12x - f21x; 
            s_fy += f12y - f21y; 
            s_fz += f12z - f21z; 

            // per-atom virial
            s_sx -= x12 * (f12x - f21x) * 0.5;
            s_sy -= y12 * (f12y - f21y) * 0.5;
            s_sz -= z12 * (f12z - f21z) * 0.5;
        }
        // save force
        g_fx[n1] = s_fx;
        g_fy[n1] = s_fy;
        g_fz[n1] = s_fz;
        // save virial
        g_sx[n1] = s_sx;
        g_sy[n1] = s_sy;
        g_sz[n1] = s_sz;
    }
}


void Fitness::find_force(void)
{
    find_force_tersoff_step1<<<Nc, MAX_ATOM_NUMBER>>>
    (
        N, Na, Na_sum, box.triclinic, num_types,
        neighbor.NN, neighbor.NL, type, ters, x, y, z, box.h, b, bp
    );
    CUDA_CHECK_KERNEL
    find_force_tersoff_step2<<<Nc, MAX_ATOM_NUMBER>>>
    (
        N, Na, Na_sum, box.triclinic, num_types, neighbor.NN, neighbor.NL, type, 
        ters, b, bp, x, y, z, box.h, pe, f12x, f12y, f12z
    );
    CUDA_CHECK_KERNEL
    find_force_tersoff_step3<<<Nc, MAX_ATOM_NUMBER>>>
    (
        N, Na, Na_sum, box.triclinic, neighbor.NN, neighbor.NL, 
        f12x, f12y, f12z, x, y, z, box.h, fx, fy, fz, sxx, syy, szz
    );
    CUDA_CHECK_KERNEL
}


